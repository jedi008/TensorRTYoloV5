#include "hip/hip_runtime.h"
#include "CudaOp.cuh"

#include <malloc.h>
#include <math.h>

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    fprintf(stderr, "addWithCuda2 is called.\n");

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    
    printf("address c: %x\n", c);
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

//blocksize ��������Ϊ2�������η�
__global__ void kernel_find_all_max_class_score(float* cuda_output, int output_box_size, int* cuda_objects_index, int* cuda_objects_index_mask,  int class_count)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int len = blockDim.x;
    const int gridsize = gridDim.x;
    const int n = bid * len + tid;
    
    __shared__ int index[128];
    index[tid] = (tid < class_count) ? tid : 0;
    __syncthreads();

    if (tid >= class_count) return;

    float* base_p = cuda_output + bid * output_box_size + 5;

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            if (base_p[index[tid]] < base_p[index[tid + offset]])
            {
                index[tid] = index[tid + offset];
            }
            __syncthreads();
        }
    }

    for (int offset = 16; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            if (base_p[index[tid]] < base_p[index[tid + offset]])
            {
                index[tid] = index[tid + offset];
            }
            __syncwarp();
        }
    }
    
    if (tid == 0)
    {
        cuda_objects_index[bid] = index[0];

        const float confidence_threshold = 0.45;
        if ((base_p[index[0]] * cuda_output[bid * output_box_size + 4]) > confidence_threshold)
        {
            cuda_objects_index_mask[bid] = 1;
            atomicAdd(cuda_objects_index_mask + gridsize, 1);
        }
        else
        {
            cuda_objects_index_mask[bid] = 0;
        }
    }
}

__global__ void kernel_init_objects(float* cuda_output, int output_box_size, float* cuda_objects, int* cuda_objects_index, int* cuda_objects_index_mask, int output_box_count)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    if (n >= output_box_count) return;

    if (!cuda_objects_index_mask[n]) return;

    int object_index = atomicAdd(cuda_objects_index_mask + output_box_count, -1) - 1;

    float* cuda_object_basep = cuda_objects + object_index * 6;
    float* cuda_output_basep = cuda_output + n * output_box_size;
    
    float pb_cx = cuda_output_basep[0];
    float pb_cy = cuda_output_basep[1];
    float pb_w = cuda_output_basep[2];
    float pb_h = cuda_output_basep[3];

    float x0 = pb_cx - pb_w * 0.5f;
    float y0 = pb_cy - pb_h * 0.5f;

    cuda_object_basep[0] = x0;
    cuda_object_basep[1] = y0;
    cuda_object_basep[2] = pb_w;
    cuda_object_basep[3] = pb_h;
    cuda_object_basep[4] = float(cuda_objects_index[n]);
    cuda_object_basep[5] = cuda_output_basep[5 + cuda_objects_index[n]] * cuda_output_basep[4];
}


__global__ void kernel_objects_sort(float* cuda_objects, int objects_count, float* cuda_objects_sorted)
{
    int tid = threadIdx.x;

    float* cuda_objects_bp = cuda_objects + tid * 6;
    float prob = cuda_objects_bp[5];
    extern __shared__ float cache[];
    cache[tid] = prob;
    __syncthreads();

    //ͳ�Ʊȵ�ǰ���Ŷȴ���м���
    int count = 0;
    for (int i = 0; i < objects_count; i++)
    {
        if (prob < cache[i])
        {
            count++;
        }
    }

    float* cuda_objects_sorted_bp = cuda_objects_sorted + count * 6;
    cuda_objects_sorted_bp[0] = cuda_objects_bp[0];
    cuda_objects_sorted_bp[1] = cuda_objects_bp[1];
    cuda_objects_sorted_bp[2] = cuda_objects_bp[2];
    cuda_objects_sorted_bp[3] = cuda_objects_bp[3];
    cuda_objects_sorted_bp[4] = cuda_objects_bp[4];
    cuda_objects_sorted_bp[5] = prob;
    //hipMemcpy(cuda_objects_sorted + count * 6, cuda_objects + tid * 6, 6 * sizeof(float), hipMemcpyDeviceToDevice); //�����ں˺���������
}


//gridsize = blocksize = objects_count�� ÿ��Box �� ���Լ������box����iou
__global__ void kernel_nms(float* cuda_objects_sorted, int objects_count, bool* dev_pickedmask, int nms_threshold, bool agnostic = false)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    //const int n = bid * blockDim.x + tid;
    if (tid == 0)
        dev_pickedmask[bid] = true;//���ó�ֵΪtrue

    float* check_box_p = cuda_objects_sorted + 6 * tid;
    extern __shared__ float areas[];
    areas[tid] = check_box_p[2] * check_box_p[3];//area
    __syncthreads();

    if (bid <= tid || !dev_pickedmask[tid]) //ִ�е���һ��ʱ�п��������̻߳�û��ִ���꣬dev_pickedmask[tid]��û�б���Ϊfalse������BUG
        return; //1.�Լ����Լ����ü���iou  2.���ں����box score������С�� ���������nmsɸ����Ч  3. �Ѿ����ų�����boxҲ�����ٿ�����

    float* box_p = cuda_objects_sorted + 6 * bid;//�����Ƿ���Ҫͨ��nms����box

    if (!agnostic && fabsf(check_box_p[4] - box_p[4]) > 0.1) return;//����ͬһ���������岻��nms



    float x1 = fmaxf(box_p[0], check_box_p[0]);
    float y1 = fmaxf(box_p[1], check_box_p[1]);
    float x2 = fminf(box_p[0] + box_p[2], check_box_p[0] + check_box_p[0]);
    float y2 = fminf(box_p[1] + box_p[3], check_box_p[1] + check_box_p[3]);
    float width = fmaxf(x2 - x1, 0);
    float height = fmaxf(y2 - y1, 0);
    float inter_area = width * height;
    float union_area = areas[bid] + areas[tid] - inter_area;
    if (inter_area / union_area > nms_threshold)
    {
        dev_pickedmask[bid] = false;
        //printf("bid tid del one: %d-%d %f %f %f %f %f %f\n", bid, tid, box_p[0], box_p[1], box_p[2], box_p[3], box_p[4], box_p[5]);
    }
}


int find_all_max_class_score(float* cuda_output, int output_box_count, float** host_objects_p)
{
    printf("find_all_max_class_score called.\n");
    int* cuda_objects_index;
    int* cuda_objects_index_mask;
    float* cuda_objects;

    //HANDLE_ERROR(hipSetDevice(0));

    float elapsed_time;
    hipEvent_t start, step1, step2, stop;
    hipEventCreate(&start);
    hipEventCreate(&step1);
    hipEventCreate(&step2);
    hipEventCreate(&stop);


    // Allocate GPU buffers.
    HANDLE_ERROR(hipMalloc((void**)&cuda_objects_index, output_box_count * sizeof(int)) );
    HANDLE_ERROR(hipMalloc((void**)&cuda_objects_index_mask, (1 + output_box_count) * sizeof(int)));//���һ��size���ڴ��array�ĺ�
    
    
    int output_box_size = 85;
    int grid_size = output_box_count;//outputBoxecount: 25200
    hipEventRecord(start);
    
    kernel_find_all_max_class_score << <grid_size, 128 >> > (cuda_output, output_box_size, cuda_objects_index, cuda_objects_index_mask, output_box_size - 5);
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipGetLastError());

    
    hipEventRecord(step1);
    hipEventSynchronize(step1);
    hipEventElapsedTime(&elapsed_time, start, step1);
    printf("find_all_max_class_score_kernel used %fms\n", elapsed_time);


    int objects_count = 0;
    HANDLE_ERROR(hipMemcpy(&objects_count, cuda_objects_index_mask + output_box_count, 1 * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMalloc((void**)&cuda_objects, 6 * objects_count * sizeof(float)));//ÿ��Boxinfo x,y,w,h,label,prob 6��ֵ
    printf("objects_count: %d\n", objects_count);
    
    hipEventRecord(step2);
    hipEventSynchronize(step2);
    hipEventElapsedTime(&elapsed_time, step1, step2);
    printf("hipMalloc cuda_objects used %fms\n", elapsed_time);



    //������ǰ����100��objects�����ò�����find_all_max_class_score_kernel �ϲ�
    kernel_init_objects << <(output_box_count + 1023)/1024, 1024 >> > (cuda_output, output_box_size, cuda_objects, cuda_objects_index, cuda_objects_index_mask, output_box_count);
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipGetLastError());



    float* cuda_objects_sorted;
    HANDLE_ERROR(hipMalloc((void**)&cuda_objects_sorted, 6 * objects_count * sizeof(float)));//ÿ��Boxinfo x,y,w,h,label,prob 6��ֵ
    kernel_objects_sort << <1, objects_count, objects_count * sizeof(float) >> > (cuda_objects, objects_count, cuda_objects_sorted);//Ĭ��objects_count���ᳬ��1024��
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipGetLastError());


    //cuda nms
    bool* cuda_pickedmask;
    HANDLE_ERROR(hipMalloc((void**)&cuda_pickedmask, objects_count * sizeof(bool)));
    kernel_nms << <objects_count, objects_count, objects_count * sizeof(float)>> > (cuda_objects_sorted, objects_count, cuda_pickedmask, 0.35);
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipGetLastError());


    bool* host_pickedmask = (bool*)malloc(objects_count * sizeof(bool));
    HANDLE_ERROR(hipMemcpy(host_pickedmask, cuda_pickedmask, objects_count * sizeof(bool), hipMemcpyDeviceToHost));
    int picked_objects_count = 0;
    for (int i = 0; i < objects_count; i++)
    {
        if (host_pickedmask[i]) ++picked_objects_count;
    }

    printf("picked_objects_count: %d\n", picked_objects_count);
    float* d_host_objects = (float*)malloc(6 * picked_objects_count * sizeof(float));
    int picked_index = 0;
    for (int i = 0; i < objects_count; i++)
    {
        if (host_pickedmask[i])
        { 
            HANDLE_ERROR(hipMemcpy(d_host_objects + picked_index * 6, cuda_objects_sorted + i * 6, 6 * sizeof(float), hipMemcpyDeviceToHost));
            ++picked_index;
        }
            
    }
    //HANDLE_ERROR(hipMemcpy(d_host_objects, cuda_objects_sorted, 6 * objects_count * sizeof(float), hipMemcpyDeviceToHost));
    //printf("1 d_host_objects: %f - %f - %f - %f - %f - %f\n", d_host_objects[0], d_host_objects[1], d_host_objects[2], d_host_objects[3], d_host_objects[4], d_host_objects[5]);
    *host_objects_p = d_host_objects;

    hipFree(cuda_objects_index);
    hipFree(cuda_objects_index_mask);
    hipFree(cuda_objects);
    hipFree(cuda_objects_sorted);
    hipFree(cuda_pickedmask);

    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("all cuda op used %fms\n", elapsed_time);

    return picked_objects_count;
}


