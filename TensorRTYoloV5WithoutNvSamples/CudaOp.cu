#include "hip/hip_runtime.h"
#include "CudaOp.cuh"


#include <stdio.h>

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

hipError_t addWithCuda2(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    fprintf(stderr, "addWithCuda2 is called.\n");

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}


__global__ void find_the_max_class_score_kernel(float* cuda_output, float* cuda_p_indexs)
{
    //int i = threadIdx.x;
    //cuda_p_indexs[0] = 0;

    int class_count = 80;
    int max_index = 0;
    for (int k = 1; k < class_count; k++)
    {
        if (cuda_output[k] > cuda_output[max_index])
        {
            max_index = k;
        }
    }
    cuda_p_indexs[0] = max_index;
}

hipError_t find_the_max_class_score(float* cuda_output, float* class_index, unsigned int size)
{
    //fprintf(stderr, "find_the_max_class_score is called.\n");
    
    float* cuda_p_indexs;
    hipError_t cudaStatus = hipSuccess;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&cuda_p_indexs, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    
    
    find_the_max_class_score_kernel << <1, 1 >> > (cuda_output, cuda_p_indexs);
    //fprintf(stderr, "cuda_p_indexs[0]: %f", cuda_p_indexs[0]); //crash!!!!!!!!!!!!!!!!!!!!



    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel in .cu!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(class_index, cuda_p_indexs, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(cuda_p_indexs);
    //hipFree(dev_a);
    //hipFree(dev_b);

    return cudaStatus;
}


//#include <math.h>
//
//int const threadsPerBlock = sizeof(unsigned long long int) * 8; // threadsPerBlock �������������unsigned longlong��λ��
//
//__device__ inline bool devIoU(float const* const a, float const* const b, const int offset, const float threshold) {
//    // ��������bbox��iou��__device__˵����cuda��ִ�У�����nms_cuda����
//    float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
//    float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
//    float width = fmaxf(right - left + offset, 0.f),
//        height = fmaxf(bottom - top + offset, 0.f);
//    float interS = width * height;
//    float Sa = (a[2] - a[0] + offset) * (a[3] - a[1] + offset);
//    float Sb = (b[2] - b[0] + offset) * (b[3] - b[1] + offset);
//    return interS > threshold * (Sa + Sb - interS);
//}
//
//__global__ void nms_cuda(const int n_boxes, const float iou_threshold, const int offset, const float* dev_boxes, unsigned long long* dev_mask) {
//    // __global__��ʾ�˺���
//    const int row_start = blockIdx.y;
//    const int col_start = blockIdx.x;
//    const int tid = threadIdx.x;
//
//    if (row_start > col_start) return; // ֻ���������ǵ�iou����
//
//    const int row_size = fminf(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
//    // ���ΪthreadsPerBlock����Ϊn_boxes���ܲ��ܱ�threadsPerBlock�������������
//    const int col_size = fminf(n_boxes - col_start * threadsPerBlock, threadsPerBlock);
//
//    __shared__ float block_boxes[threadsPerBlock * 4];
//    // �����ڴ棬��ͬһ�߳̿���Ƶ�����ʵ�64��bbox����Ϣ�ŵ������ڴ�
//    // �����ڴ��ͬһ�߳̿��е������ڴ湲��
//    // ����ÿ���̣߳������һ��bbox�ŵ������ڴ��� 
//    if (tid < col_size) {
//        block_boxes[tid * 4 + 0] = dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 0];
//        // dev_boxes��һά������׵�ַ
//        block_boxes[tid * 4 + 1] = dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 1];
//        block_boxes[tid * 4 + 2] = dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 2];
//        block_boxes[tid * 4 + 3] = dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 3];
//    }
//    __syncthreads(); // ͬ����ʹ�ù����ڴ�һ��Ҫͬ������64���̰߳�bbox�ŵ������ڴ���ټ�������iou
//    //������ VS��ʶ�𣬵���cuda��ʶ�𣬿�������
//
//    if (tid < row_size) {
//        const int cur_box_idx = threadsPerBlock * row_start + tid;
//        const float* cur_box = dev_boxes + cur_box_idx * 4;
//        int i = 0;
//        unsigned long long int t = 0;
//        int start = 0;
//        if (row_start == col_start) {
//            start = tid + 1; // �Խ����ϵ�����������iou
//        }
//        for (i = start; i < col_size; i++) {
//            // ÿ���߳�Ҫ��col_size��iou
//            if (devIoU(cur_box, block_boxes + i * 4, offset, iou_threshold)) {
//                t |= 1ULL << i; // ���iou������ֵ��ͨ��λ���㣬tΪ64λ0 or 1����t�ĵ�iλ��Ϊ1
//            }
//        }
//        dev_mask[cur_box_idx * gridDim.y + col_start] = t; // �޸�mask[cur_box_idx][col_start]Ϊt
//    }
//}
